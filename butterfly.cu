#include "hip/hip_runtime.h"
#ifndef BUTTERFLY_CU
#define BUTTERFLY_CU 1

#include "butterfly.h"
#include <hip/hip_runtime.h>
#include<hip/hip_runtime.h>

#define blocksize 4
//matrix must be mutiple of 2 in size 

struct data {
	static int vals[4][3];
	int bigindex[4];
};
data * writeloc(int rowsize, int bsize){
	// initilize the vals and big index 
	//write them to GPU mem
	data cur;
	cur.vals[0][0] = 1;
	cur.vals[0][1] = 1; // c1  m 3 
	cur.vals[0][2] = 1;

	cur.vals[1][0] = -1;
	cur.vals[1][1] = 1;
	cur.vals[1][2] = -1;

	cur.vals[2][0] = 1;
	cur.vals[2][1] = -1;
	cur.vals[2][2] = -1;

	cur.vals[3][0] = -1;
	cur.vals[3][1] = -1;
	cur.vals[3][2] = 1;

	cur.bigindex[0] = 0;  // index of each block in M and write 
	cur.bigindex[1] = bsize / 2;
	cur.bigindex[2] = rowsize * bsize / 2;
	cur.bigindex[3] = rowsize * bsize / 2 + bsize / 2;
	data * gpuloc;
	hipMalloc((void**)&gpuloc, sizeof(data));
	hipMemcpy(gpuloc, & cur, sizeof(data), hipMemcpyHostToDevice);
	return gpuloc;

}

__global__ void gpu_buttermulti(double * C, int bsize, int rowsize, bint * A, bint * M, bint * B) {

	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	

}



/*

void blockBmidd(double * C, int bsize, int rowsize, bint * A, bint * M, bint * B){
	//A M B and C must be pointers into the GPU memory 
	//C is the location in a rowsize * row size matrix where the block of bsize * bsize will start 
	//A B are the correct entries in 2 butterfly matrices to start this block
	//M is the start in 
	//compute chunks of C 
	// C must be initilized to a row size by row size matrix 
	assert(bsize <= rowsize);
	assert(rowsize % rowsize == 0);
	//by a i mean m 
	//0 c1 = a11 + a12 +a21 +a22    1 c2 = a11 - a12 +a21 - a22
	//2 c3 = a11 + a12 -a21 - a22	3 c4 = a11 - a12 - a21 +a22
	static int vals[4][3];
	


	// each thread will write 1 entries of c by loading 1 entires of A into local mem
	//each block will load a tile from each of the 4 quadrants 




	//start parrallel section 
	for (int block = 0; block < 4; block++){//  split matrix into blocks 

		//	printf("inddex:%d   block:%d\n", index[block], block);
		for (int row = 0; row < bsize / 2; row++){//itterate down to next row
			for (int col = 0; col < bsize / 2; col++){//itterate accross the row 
				//set intital value by loading upper left corrner
				
				C[bigindex[block] + row* rowsize + col] =
					M[row * rowsize + col];
				for (int j = 1; j < 4; j++){

					C[bigindex[block] + row* rowsize + col] +=
						M[bigindex[j] + row* rowsize + col] * vals[block][j - 1];
				}

			}
		}
	}

	for (int row = 0; row < bsize; row++){
		for (int col = 0; col < bsize; col++){
			C[row* rowsize + col] *= A[row];
		}
	}

	//f1 = d1*  r'0 			f2 = d2 * r'1  	
	//f3 = d3 * r'0 			f4 = d4 * r'1  
	//devide by 2 
	for (int row = 0; row < bsize; row++){
		for (int col = 0; col < bsize; col++){
			C[row* rowsize + col] *= B[col] * .5;
		}
	}
	//end parrallel section 
}
*/



matrix middlebmulti(butterfly a, matrix m, butterfly b){
	//compute chunks of C 
	assert(a.size = m.n);
	assert(b.depth = a.depth);
	// check gpu ready



	// --------------------SET PARAMETERS AND DATA -----------------------
	int gpucount = 0;
	hipEvent_t start, stop; // using cuda events to measure time
	float elapsed_time_ms; // which is applicable for asynchronous code also
	hipError_t errorcode;

	errorcode = hipGetDeviceCount(&gpucount);
	if (errorcode == hipErrorNoDevice) {
		printf("No GPUs are visible\n");
		exit(-1);
	}


	double * Agpu, * Mgpu, * Bgpu , * Cgpu, *Dgpu;
	int butterSize = a.depth * a.size * sizeof(bint);
	int matSize = m.n * m.n * sizeof(double);
	// push a m b onto gpu 
	hipMalloc((void**)&Agpu, butterSize);
	hipMalloc((void**)&Bgpu, butterSize);
	hipMalloc((void**)&Agpu, matSize);

	hipMemcpy(Agpu, a.entries, butterSize, hipMemcpyHostToDevice);
	hipMemcpy(Bgpu, b.entries, butterSize, hipMemcpyHostToDevice);
	hipMemcpy(Mgpu, m.body, matSize, hipMemcpyHostToDevice);

	dim3 Grid(m.n / blocksize , m.n / blocksize); //Grid structure
	dim3 Block(blocksize, blocksize); //Block structure

	matrix C(m.n, true);
	//start at lowest depth and work outward
	if (a.depth == 2){
		
		//cuda alloc spae for D
		matrix D(m.n, true);
		hipMalloc((void**)&Dgpu, matSize);
		//set vars 
// reset internal values 
		//upper left  a1   m11  b1
		gpu_buttermulti<<<Grid, Block>>>(Dgpu, m.n / 2, m.n,
			Agpu + m.n, Mgpu, Bgpu + m.n);
		
		//upper right   a1  m12   b2
		gpu_buttermulti << <Grid, Block >> >(Dgpu + m.n / 2, m.n / 2, m.n,
			Agpu + m.n, Mgpu + m.n / 2, Bgpu + m.n / 2 + m.n);

		//lower left  a2   m21  b1
		gpu_buttermulti << <Grid, Block >> >(Dgpu + m.n * (m.n) / 2, m.n / 2, m.n, 
			Agpu + m.n + m.n / 2, Mgpu + m.n * m.n / 2, Bgpu + m.n);

		//lower right  a2   m22  b2
		gpu_buttermulti << <Grid, Block >> >(Dgpu + m.n * (m.n + 1) / 2,
			m.n / 2, m.n, Agpu + m.n + m.n / 2,
			Mgpu + m.n * (m.n + 1) / 2, Bgpu + m.n + m.n / 2);


		
		//make space for C on gpu
		hipMalloc((void**)&Cgpu, matSize);
		// now the depth 1 butterfly
		gpu_buttermulti << <Grid, Block >> >(Cgpu, m.n, m.n, Agpu, Dgpu, Bgpu);
		hipMemcpy(C.body, Cgpu, matSize, hipMemcpyDeviceToHost);

		free(D.body);
		hipFree(Dgpu);

	}
	else {
		hipMalloc((void**)&Cgpu, matSize);
		//make space for C on gpu
		gpu_buttermulti << <Grid, Block >> >(Cgpu, m.n, m.n, Agpu, Mgpu, Bgpu);
		hipMemcpy(C.body, Cgpu, matSize, hipMemcpyDeviceToHost);
	}


	hipFree(Cgpu);
	hipFree(Mgpu);
	hipFree(Agpu);
	hipFree(Bgpu);
	return C;

}
void blockBleft(bint * C, int bsize, int rowsize, bint * A, bint * M){


	// c is   c0   A0 + a2		c1  a1 + a3
	//		  c2   a0 - a2      c3  a1 -a3

	//fill pairs in a colomns at a time
	for (int row = 0; row < bsize / 2; row++){//itterate down to next row
		for (int col = 0; col < bsize; col++){//itterate accross the row 
			C[row* rowsize + col] = M[row* rowsize + col];
			C[(row + bsize / 2)* rowsize + col] = M[row* rowsize + col];

			C[row * rowsize + col] += M[(row + bsize / 2)* rowsize + col];
			C[(row + bsize / 2)* rowsize + col] -= M[(row + bsize / 2)* rowsize + col];

		}
	}

	//C.printMatrix();
	//r0 r1 are diagonals of a 
	//d1 =  r0  * c1     d2 =  r0 * c2
	//d3 =  r1	* c3	d4 =  r1  * c4
	for (int row = 0; row < bsize; row++){
		for (int col = 0; col < bsize; col++){
			C[row* rowsize + col] *= A[row] / sqrt(2.0);
		}
	}

}
matrix leftbmulti(butterfly b, matrix m){
	matrix C(m.n, false);


	if (b.depth == 2){
		matrix D(m.n, true);
		//upper left  a1   m11  b1
		blockBleft(D.body, m.n / 2, m.n, b.entries + m.n, m.body);
		//upper right   a1  m12   b2
		blockBleft(D.body + m.n / 2, m.n / 2, m.n, b.entries + m.n, m.body + m.n / 2);
		//lower left  a2   m21  b1
		blockBleft(D.body + m.n * (m.n) / 2, m.n / 2, m.n, b.entries + m.n + m.n / 2,
			m.body + m.n * m.n / 2);

		//lower right  a2   m22  b2
		blockBleft(D.body + m.n * (m.n + 1) / 2,
			m.n / 2, m.n, b.entries + m.n + m.n / 2,
			m.body + m.n * (m.n + 1) / 2);
		//	printf("\nD is \n");
		//	D.printMatrix();
		// now the depth 1 butterfly
		blockBleft(C.body, m.n, m.n, b.entries, D.body);

		free(D.body);

	}
	else blockBleft(C.body, m.n, m.n, b.entries, m.body);
	return C;

}


#endif