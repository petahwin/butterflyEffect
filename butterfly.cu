#include "hip/hip_runtime.h"
#include "butterfly.h"
#include <hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#define blocksize 8
#define MaxTW 8
//matrix must be mutiple of 2 in size 


/*----------------------------Matrix METHODS---------------------------*/

Matrix::Matrix(int n_in, bool randfill) {
	n = n_in;
	if (randfill == true) {
		body = (double *)malloc(sizeof(double)* n* n);
		for (int i = 0; i < n* n; i++) body[i] = (double) rand();
	}
	else {
		body = (double *)calloc(n* n, sizeof(double));
	}
}

void Matrix::printMatrix(void) {
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			printf("%g    ", body[i*n + j]);
		}
		printf("\n\n");
	}
}

//assume m is correct 
void Matrix::percenterror(Matrix m, Matrix A){
	assert(m.n == A.n);
	double total = 0.0;
	double totalerr = 0.0;
	for (int i = 0; i < m.n; i++){
		for (int j = 0; j < m.n; j++){
			total += m.body[i*m.n + j];
			totalerr += abs(m.body[i*m.n + j] - A.body[i*m.n + j]);
		}

	}
	double percenterr = totalerr / total;

	printf("\ntotal error was %g  percent error \
		           was %g\n\n", totalerr, percenterr);
	return;
}

/*--------------------------Butterfly METHODS--------------------------*/

Butterfly::Butterfly(int insize, int indepth) {
	size = insize;
	depth = indepth;
	entries = (bint *)malloc(depth * size * sizeof(bint));
	transposed = false;

	int r = rand();
	for (int i = 0; i < indepth * insize; i++){
		entries[i] =  (bint)rand();// / INT_MAX;
	}
	return;
}

//very cheep function for transposing  
void Butterfly::transpose(void) {
	transposed = !transposed;
}

void Butterfly::printEntries(void) {
	for (int i = 0; i < size * depth; i++) {
		printf("%g  ", entries[i]);
	}
}

struct Data {
    int vals[4][3];
	int bigindex[4];
};

Data * writeloc(int rowsize, int bsize){
	// initilize the vals and big index 
	//write them to GPU mem
	Data cur;
	cur.vals[0][0] = 1;
	cur.vals[0][1] = 1; // c1  m 3 
	cur.vals[0][2] = 1;

	cur.vals[1][0] = -1;
	cur.vals[1][1] = 1;
	cur.vals[1][2] = -1;

	cur.vals[2][0] = 1;
	cur.vals[2][1] = -1;
	cur.vals[2][2] = -1;

	cur.vals[3][0] = -1;
	cur.vals[3][1] = -1;
	cur.vals[3][2] = 1;

	cur.bigindex[0] = 0;  // index of each block in M and write 
	cur.bigindex[1] = bsize / 2;
	cur.bigindex[2] = rowsize * bsize / 2;
	cur.bigindex[3] = rowsize * bsize / 2 + bsize / 2;
	printf("big 1:%d  2:%d   3:%d   4:%d\n", cur.bigindex[0], cur.bigindex[1], cur.bigindex[2], cur.bigindex[3]);
	Data * gpuloc;
	hipMalloc((void**)&gpuloc, sizeof(Data));
	hipMemcpy(gpuloc, & cur, sizeof(Data), hipMemcpyHostToDevice);
	return gpuloc;

}

__global__ void gpu_buttermulti(double * C, int bsize, int rowsize, bint * A, double * M, bint * B, Data * data) {
	//do the row and colith entry in in quadrent 
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;
	double Clocal[4];
	for (int block = 0; block < 4; block++){
		Clocal[block] = M[row * rowsize + col];// row * rowsize + col;
	//C[ row* rowsize + col] = row * rowsize + col;
			//M[row * rowsize + col];
		for (int j = 1; j < 4; j++){

			Clocal[block] +=
				M[data->bigindex[j] + row* rowsize + col] * data->vals[block][j - 1];
		}
	}
	for (int block = 0; block < 4; block++){
		Clocal[block] *= A[row + bsize / 2 * (block / 2)] *
							B[col + (block % 2) * bsize / 2] * .5;
	}
	for (int block = 0; block < 4; block++){
		C[data->bigindex[block] + row* rowsize + col] = Clocal[block];
	}

}
__global__ void gpu_LeftButtermulti(double * C, int bsize, int rowsize, bint * A, double * M) {
	//each responsible for 4 entries 
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;
	// c is   c0   A0 + a2		c1  a1 + a3
	//		  c2   a0 - a2      c3  a1 -a3
	double Clocal[2];
	Clocal[0]= M[row* rowsize + col];
	Clocal[1]= M[row* rowsize + col];

	Clocal[0]+= M[(row + bsize / 2)* rowsize + col];
	Clocal[1] -= M[(row + bsize / 2)* rowsize + col];

	Clocal[0]*= A[row] / sqrt(2.0);
	Clocal[1]*= A[row + bsize / 2] / sqrt(2.0);
	C[row* rowsize + col] = Clocal[0];
	C[(row + bsize / 2)* rowsize + col] = Clocal[1];


}



Matrix middlebmulti(Butterfly a, Matrix m, Butterfly b){
	//compute chunks of C 
	assert(a.size = m.n);
	assert(b.depth = a.depth);
	// check gpu ready



	// --------------------SET PARAMETERS AND DATA -----------------------
	int gpucount = 0;
	hipError_t errorcode;

	errorcode = hipGetDeviceCount(&gpucount);
	if (errorcode == hipErrorNoDevice) {
		printf("No GPUs are visible\n");
		exit(-1);
	}


	double * Agpu, * Mgpu, * Bgpu , * Cgpu, *Dgpu;
	int butterSize = a.depth * a.size * sizeof(bint);
	int matSize = m.n * m.n * sizeof(double);
	// push a m b onto gpu 
	hipMalloc((void**)&Agpu, butterSize);
	hipMalloc((void**)&Bgpu, butterSize);
	hipMalloc((void**)&Mgpu, matSize);

	hipMemcpy(Agpu, a.entries, butterSize, hipMemcpyHostToDevice);
	hipMemcpy(Bgpu, b.entries, butterSize, hipMemcpyHostToDevice);
	hipMemcpy(Mgpu, m.body, matSize, hipMemcpyHostToDevice);

	dim3 Grid(m.n / (blocksize * 2), m.n / (blocksize * 2)); //Grid structure
	dim3 Block(blocksize, blocksize); //Block structure
	printf("\nblock:%d grid:%d m.n:%d \n\n ", blocksize, m.n / (blocksize * 2), m.n);
	Matrix C(m.n, true);
	//start at lowest depth and work outward
	if (a.depth == 2){
		dim3 miniGrid(m.n / (blocksize * 4), m.n / (blocksize * 4)); //Grid structure
		//cuda alloc spae for D
		Matrix D(m.n, true);
		hipMalloc((void**)&Dgpu, matSize);
		//set vars 
		Data * data = writeloc(m.n, m.n / 2);

		//upper left  a1   m11  b1
		gpu_buttermulti<<<miniGrid, Block>>>(Dgpu, m.n / 2, m.n,
			Agpu + m.n, Mgpu, Bgpu + m.n, data);


		//upper right   a1  m12   b2
		gpu_buttermulti << <miniGrid, Block >> >(Dgpu + m.n / 2, m.n / 2, m.n,
			Agpu + m.n, Mgpu + m.n / 2, Bgpu + m.n / 2 + m.n, data);

		//lower left  a2   m21  b1
		gpu_buttermulti << <miniGrid, Block >> >(Dgpu + m.n * (m.n) / 2, m.n / 2, m.n, 
			Agpu + m.n + m.n / 2, Mgpu + m.n * m.n / 2, Bgpu + m.n, data);


		//lower right  a2   m22  b2
		gpu_buttermulti << <miniGrid, Block >> >(Dgpu + m.n * (m.n + 1) / 2,
			m.n / 2, m.n, Agpu + m.n + m.n / 2,
			Mgpu + m.n * (m.n + 1) / 2, Bgpu + m.n + m.n / 2, data);
		hipFree(data);

		data = writeloc(m.n, m.n );
		//make space for C on gpu
		hipMalloc((void**)&Cgpu, matSize);
		// now the depth 1 Butterfly
		gpu_buttermulti << <Grid, Block >> >(Cgpu, m.n, m.n, Agpu, Dgpu, Bgpu, data);

		hipMemcpy(C.body, Cgpu, matSize, hipMemcpyDeviceToHost);

		free(D.body);
		hipFree(Dgpu);

	}
	else {
		Data * data = writeloc(m.n, m.n);
		hipMalloc((void**)&Cgpu, matSize);
		//make space for C on gpu
		gpu_buttermulti << <Grid, Block >> >(Cgpu, m.n, m.n, Agpu, Mgpu, Bgpu, data);
		hipMemcpy(C.body, Cgpu, matSize, hipMemcpyDeviceToHost);
		hipFree(data);
	}


	hipFree(Cgpu);
	hipFree(Mgpu);
	hipFree(Agpu);
	hipFree(Bgpu);
	return C;

}

Matrix leftbmulti(Butterfly a, Matrix m){
	//compute chunks of C 
	assert(a.size = m.n);
	// check gpu ready



	// --------------------SET PARAMETERS AND DATA -----------------------
	int gpucount = 0;
	hipError_t errorcode;

	errorcode = hipGetDeviceCount(&gpucount);
	if (errorcode == hipErrorNoDevice) {
		printf("No GPUs are visible\n");
		exit(-1);
	}


	double * Agpu, *Mgpu, *Cgpu, *Dgpu;
	int butterSize = a.depth * a.size * sizeof(bint);
	int matSize = m.n * m.n * sizeof(double);
	// push a m b onto gpu 
	hipMalloc((void**)&Agpu, butterSize);
	
	hipMalloc((void**)&Mgpu, matSize);

	hipMemcpy(Agpu, a.entries, butterSize, hipMemcpyHostToDevice);

	hipMemcpy(Mgpu, m.body, matSize, hipMemcpyHostToDevice);

	dim3 Grid(m.n / (blocksize ), m.n / (blocksize * 2)); //Grid structure
	dim3 Block(blocksize, blocksize); //Block structure
	printf("\nblock:%d grid:%d m.n:%d \n\n ", blocksize, m.n / (blocksize * 2), m.n);
	Matrix C(m.n, true);
	//start at lowest depth and work outward
	if (a.depth == 2){
		dim3 miniGrid(m.n / (blocksize * 2), m.n / (blocksize * 4)); //Grid structure
		//cuda alloc spae for D
		Matrix D(m.n, true);
		hipMalloc((void**)&Dgpu, matSize);
		
		// reset internal values 

		//upper left  a1   m11  b1
		gpu_LeftButtermulti << <miniGrid, Block >> >(Dgpu, m.n / 2, m.n,
			Agpu + m.n, Mgpu );


		//upper right   a1  m12   b2
		gpu_LeftButtermulti << <miniGrid, Block >> >(Dgpu + m.n / 2, m.n / 2, m.n,
			Agpu + m.n, Mgpu + m.n / 2);

		//lower left  a2   m21  b1
		gpu_LeftButtermulti << <miniGrid, Block >> >(Dgpu + m.n * (m.n) / 2, m.n / 2, m.n,
			Agpu + m.n + m.n / 2, Mgpu + m.n * m.n / 2);


		//lower right  a2   m22  b2
		gpu_LeftButtermulti << <miniGrid, Block >> >(Dgpu + m.n * (m.n + 1) / 2,
			m.n / 2, m.n, Agpu + m.n + m.n / 2,
			Mgpu + m.n * (m.n + 1) / 2);
		

		
		//make space for C on gpu
		hipMalloc((void**)&Cgpu, matSize);
		// now the depth 1 Butterfly
		gpu_LeftButtermulti << <Grid, Block >> >(Cgpu, m.n, m.n, Agpu, Dgpu);

		hipMemcpy(C.body, Cgpu, matSize, hipMemcpyDeviceToHost);

		free(D.body);
		hipFree(Dgpu);

	}
	else {
		
		hipMalloc((void**)&Cgpu, matSize);
		//make space for C on gpu
		gpu_LeftButtermulti << <Grid, Block >> >(Cgpu, m.n, m.n, Agpu, Mgpu);
		hipMemcpy(C.body, Cgpu, matSize, hipMemcpyDeviceToHost);
		
	}


	hipFree(Cgpu);
	hipFree(Mgpu);
	hipFree(Agpu);
	
	return C;

}